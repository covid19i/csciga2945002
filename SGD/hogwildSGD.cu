#include "hip/hip_runtime.h"
//
//  cudaSGD.cpp
//  parallelSGD
//
//  Created by Yue Sun on 4/22/20.
//  Copyright © 2020 Yue Sun. All rights reserved.
//


#include "dataReader.h"
#include "PSGD.h"
#include "MultiLog.h"
#include "LossType.h"
#include "hogwild.h"

#include <iostream>
#include <cstdlib>
#include <string>
#include <random>
#include <chrono>
#include <omp.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>


using namespace std;
typedef unsigned char uchar;

//Should probably be 800 (= 785/32 * 32) for MNIST job

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}


// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

int main(int argc, const char * argv[]) {
    printf("Started main.\n");
    mnist data;
    int n_images;
    int size_image;
    double **tempData;
    tempData = data.read_mnist_images("train-images.idx3-ubyte",n_images, size_image);
    //n_images 60000,size_image=784
    double * trainingData;
    hipHostMalloc((void**)&trainingData,n_images*(size_image+1)*sizeof(double));
    Check_CUDA_Error("Malloc trainingData Host failed");
    
    for(int i=0;i<n_images;i++){
        for(int j=0;j<size_image+1;j++){
            trainingData[i*(size_image+1)+j]=tempData[i][j];
        }
    }
    printf("trainingData[24356][%d] = %.10f\n", QUERY, tempData[24356][QUERY]);
    int n_labels;
    uchar *tempLabel;
    
    tempLabel = data.read_mnist_labels("train-labels.idx1-ubyte",n_labels);
    uchar* trainingLabel;
    hipHostMalloc((void**)&trainingLabel,n_labels*sizeof(uchar));
    for(int i=0;i<n_labels;i++){
        trainingLabel[i]=tempLabel[i];
    }
    printf("trainingLabel[24356] = %u\n", tempLabel[24356]);
   
    int n_images_test;
    int size_image_test;
    double **testingData;
    testingData = data.read_mnist_images("t10k-images-idx3-ubyte", n_images_test, size_image_test);
    int n_labels_test;
    uchar *testingLabels;
    testingLabels = data.read_mnist_labels("t10k-labels-idx1-ubyte",n_labels_test);
    printf("Data loaded to Host.\n");

    double *trainingData_d;
    uchar  *trainingLabel_d;
    hipMalloc(&trainingData_d, (n_images * (size_image+1) *sizeof(double)));
    Check_CUDA_Error("Malloc trainingData Device failed");
    hipMalloc(&trainingLabel_d, n_images *sizeof(uchar));
    hipMemcpy(trainingData_d, trainingData, (n_images * (size_image+1) *sizeof(double)), hipMemcpyHostToDevice);
    Check_CUDA_Error("Copy to trainingData Device failed");
    hipMemcpy(trainingLabel_d, trainingLabel,n_images *sizeof(uchar), hipMemcpyHostToDevice);
    hipDeviceSynchronize(); 
    printf("Data loaded to device.\n");

    PSGD psgd(1);
    psgd.initialize(size_image+1,10);
    //vector<double> weight_openmp = psgd.getWeight();
    
    
    
    double* weight;
    int weight_size=(size_image+1)*10;
    hipHostMalloc((void**)&weight,weight_size*sizeof(double));
 
    //initialize the weight
    int seed =1;//chrono::system_clock::now().time_since_epoch().count();
    default_random_engine generator (seed);
    normal_distribution<double> distribution (0.0,1.0);
    for (int i=0;i<weight_size;i++){
        weight[i]=0;//distribution(generator);
    }
    //psgd.testGPU(weight, tempData, tempLabel, n_images, size_image+1, 10);
    psgd.testGPU(weight, testingData, testingLabels, n_images_test, size_image+1, 10);
    
    double* weight_d;
    hipMalloc(&weight_d, weight_size*sizeof(double));
    Check_CUDA_Error("Malloc Weights Device failed");
    hipMemcpy(weight_d, weight, weight_size*sizeof(double), hipMemcpyHostToDevice);
    printf("weight[%d] = %.10f\n", QUERY, weight[QUERY]);
    hipDeviceSynchronize();    /*
    printf("\nGPU:\n");
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) { 
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
    }*/
    
    printf("Enter iterations (> 10):\n");
    int n_iterations=10;
    scanf("%d", &n_iterations);
    
    double eta;
    eta=0.001;
    //printf("\nEnter learning rate (eta = 0.001):\n");
    //scanf("%lf", &eta);
    
    double lambda;
    lambda=0.001;
    //printf("\nEnter regularization parameter (lambda = 0.001):\n");
    //scanf("%lf", &lambda);
    
    double oldLoss=getLoss(weight,tempData,tempLabel,n_images,size_image+1,10,lambda);
    printf("old loss: %f \n",oldLoss);
    double t = omp_get_wtime();
          printf("weight[%d] = %f\n", QUERY, weight[QUERY]);

    //printf("Enter number of blocks to run simultaneously. Each block will process one data point at any time.\n");
    int n_blocks = 16;
    //scanf("%d", n_blocks);
    //update the weight
    for(long j=0;j<n_iterations;j++){
	run_hogwild_one_processor<<<n_blocks, BLOCK_SIZE>>>(weight_d,trainingData_d,trainingLabel_d,eta,n_images,size_image+1,10,lambda, j, n_iterations);
        Check_CUDA_Error("Kernel Failed to launch\n");
	hipDeviceSynchronize();
        //printf("Iteration %d done.\n", j);
        if(j < 2){
          hipMemcpy(weight, weight_d, weight_size*sizeof(double), hipMemcpyDeviceToHost);
	  hipDeviceSynchronize();
          printf("Iteration no: %d, weight[%d] = %.10f\n", j, QUERY, weight[QUERY]);
        }
    	if(j %(n_iterations/5) == 0 || j == n_iterations-1 || j == 10 || j == 100 || j == 1000 || j == 10000 || j == 100000 || j == 1000000 || j == 10000000){
          hipMemcpy(weight, weight_d, weight_size*sizeof(double), hipMemcpyDeviceToHost);
	  hipDeviceSynchronize();
          printf("Iteration no: %d, weight[%d] = %.10f\n", j, QUERY, weight[QUERY]);
	  double loss_now = getLoss(weight, tempData, tempLabel, n_images, size_image+1,10, lambda);
	  printf("Training (log)loss: %.10f\t thread:%d\n",loss_now, omp_get_thread_num());
	  psgd.testGPU(weight, tempData, tempLabel, n_images, size_image+1, 10);
	  psgd.testGPU(weight, testingData, testingLabels, n_images_test, size_image+1, 10);
	}
    }
    
    t = omp_get_wtime() - t;
    printf("\nTime elapsed in training = %f sec\n", t);
    printf("Time elapsed in training per iteration = %f sec\n", t/n_iterations);
    hipMemcpy(weight, weight_d, weight_size*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    double newLoss=getLoss(weight,tempData,tempLabel,n_images,size_image+1,10,lambda);
    printf("new loss: %f \n",newLoss);

    MultiLog mlog;
    mlog.setLambda(lambda);//Regularization parameter
    psgd.loss = &mlog;//setting the loss function in PSGD object
    printf("\nNo of iterations for each thread block: %d\n", n_iterations);
    printf("No of threads in each block: %d\n", BLOCK_SIZE);
    printf("No of blocks: %d\n", n_blocks);
    printf("Lambda (Regularization Parameter): %lf\n", mlog.getLambda());
    printf("Eta (Learning Rate): %lf\n", eta);

    psgd.testGPU(weight, testingData, testingLabels, n_images_test, size_image+1, 10);
 
    printf("End\n");
    free(tempData);
    free(tempLabel);
    checkCuda( hipFree(trainingData));
    hipFree(trainingLabel);
    hipFree(weight);
	checkCuda( hipFree(trainingData_d));	
	checkCuda( hipFree(trainingLabel_d));	
	checkCuda( hipFree(weight_d));
}

